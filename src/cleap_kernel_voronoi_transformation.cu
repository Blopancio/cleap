#include "hip/hip_runtime.h"
#ifndef _CLEAP_KERNEL_VORONOI_TRANSFORMATION_H
#define _CLEAP_KERNEL_VORONOI_TRANSFORMATION_H

template<unsigned int block_size>
__global__ void cleap_kernel_circumcenter_calculus( float4* vertex_data, GLuint* triangles, float4* circumcenters, int face_count){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if( i<face_count ){
        float4 a, b, c;
        float3 A, B, C, u, v, r1, r2;
        float factor;


        a = vertex_data[triangles[i*3]];
        b = vertex_data[triangles[i*3+1]];
        c = vertex_data[triangles[i*3+2]];

        A = make_float3(b.x - a.x, b.y - a.y, b.z - a.z);
        B = make_float3(c.x - b.x, c.y - b.y, c.z - b.z);
        C = make_float3(a.x - c.x, a.y - c.y, a.z - c.z);

        u = cleap_d_cross_product(A,B);
        u = make_float3(u.x, -u.y, u.z);

        v = cleap_d_cross_product(C,u);
        v = make_float3(v.x, -v.y, v.z);

        r1 = make_float3(c.x + a.x, c.y + a.y, c.z + a.z);
        r1 = make_float3(r1.x/2.0, r1.y/2.0, r1.z/2.0);

        u = make_float3(u.x/2.0, u.y/2.0, u.z/2.0);

        factor = cleap_d_magnitude(u);
        factor = cleap_d_dot_product(A, B) / (8* factor*factor);

        r2 = make_float3(factor*v.x, factor*v.y, factor*v.z);

        circumcenters[i] = make_float4(r1.x + r2.x, r1.y + r2.y, r1.z + r2.z, 1.0);
    }
}

#endif