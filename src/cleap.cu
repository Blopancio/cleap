#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////
//                                                                           	//
//	cleap                                                                   //
//	A library for handling / processing / rendering 3D meshes.	        //
//                                                                           	//
//////////////////////////////////////////////////////////////////////////////////
//										//
//	Copyright © 2011 Cristobal A. Navarro.					//
//										//	
//	This file is part of cleap.						//
//	cleap is free software: you can redistribute it and/or modify		//
//	it under the terms of the GNU General Public License as published by	//
//	the Free Software Foundation, either version 3 of the License, or	//
//	(at your option) any later version.					//
//										//
//	cleap is distributed in the hope that it will be useful,		//
//	but WITHOUT ANY WARRANTY; without even the implied warranty of		//
//	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the	    	//
//	GNU General Public License for more details.				//
//										//
//	You should have received a copy of the GNU General Public License	//
//	along with cleap.  If not, see <http://www.gnu.org/licenses/>. 		//
//										//
//////////////////////////////////////////////////////////////////////////////////



// public headers
#include "cleap_private.h"
#include "cleap_config.h"

// important cuda runtime headers
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// kernel headers
#include "cleap_kernel_utils.cu"
#include "cleap_kernel_normalize_normals.cu"
#include "cleap_kernel_delaunay_transformation.cu"
#include "cleap_kernel_paint_mesh.cu"

// context creation header for opengl
// linux
#include "cleap_glx_context.cu"

#include <math.h>  
// default blocksize
int CLEAP_CUDA_BLOCKSIZE = 256;

// timer structures
struct timeval t_ini, t_fin;

// cleap author
char CLEAP_AUTHOR[] = "Cristobal A. Navarro";

// cuda textures
texture<GLuint, 1, hipReadModeElementType> tex_triangles;
texture<int, 1, hipReadModeElementType> tex_edges;

int cleap_mesh_is_wireframe(_cleap_mesh *m){
	return m->wireframe;
}
int cleap_mesh_is_solid(_cleap_mesh *m){
	return m->solid;
}
void cleap_mesh_set_wireframe(_cleap_mesh *m, int w){
	m->wireframe = w;
}
void cleap_mesh_set_solid(_cleap_mesh *m, int s){
	m->solid = s;
}

float cleap_get_bsphere_r(_cleap_mesh *m){

        float view_diamx = m->max_coords.x - m->min_coords.x;
        float view_diamy = m->max_coords.y - m->min_coords.y;
        float view_diamz = m->max_coords.z - m->min_coords.z;
	return 0.5f*sqrt(powf(view_diamx, 2) + powf(view_diamz, 2) + powf(view_diamy, 2));
}
float cleap_get_bsphere_x(_cleap_mesh *m){

	return	0.5f*(m->max_coords.x + m->min_coords.x);
}
float cleap_get_bsphere_y(_cleap_mesh *m){

	return	0.5f*(m->max_coords.y + m->min_coords.y);
}
float cleap_get_bsphere_z(_cleap_mesh *m){

	return	0.5f*(m->max_coords.z + m->min_coords.z);
}


CLEAP_RESULT cleap_init(){

	_cleap_print_splash();
	_cleap_init_cuda();

	return CLEAP_SUCCESS;

}

CLEAP_RESULT cleap_init_no_render(){

	//_cleap_print_splash();
	_cleap_create_glx_context();
	_cleap_init_glew();
	_cleap_init_cuda();

	return CLEAP_SUCCESS;
}

CLEAP_RESULT cleap_end(){
	_cleap_destroy_glx_context();
	return CLEAP_SUCCESS;
}

int cleap_get_vertex_count(_cleap_mesh *hm){
	return hm->vertex_count;
}

int cleap_get_edge_count(_cleap_mesh *hm){
	return hm->edge_count;
}

int cleap_get_face_count(_cleap_mesh *hm){
	return hm->face_count;
}

_cleap_mesh* cleap_load_mesh(const char* filename){

	_cleap_mesh *m = new _cleap_mesh();	// create mew mesh
	_cleap_host_load_mesh(m, filename);	// load host part
	_cleap_device_load_mesh(m);		// load device part

	return m;
}

CLEAP_RESULT cleap_paint_mesh(_cleap_mesh *m, GLfloat r, GLfloat g, GLfloat b, GLfloat a ){

	//printf("CLEAP::kernel::paint_mesh::");
	size_t bytes;
	float4 *dptr;
	int vcount = cleap_get_vertex_count(m);
	cleap_device_mesh *dm = m->dm;
	hipGraphicsMapResources(1, &dm->vbo_c_cuda, 0);
	hipGraphicsResourceGetMappedPointer((void **)&dptr, &bytes, dm->vbo_c_cuda);

	dim3 dimBlock(CLEAP_CUDA_BLOCKSIZE);
	dim3 dimGrid((vcount+CLEAP_CUDA_BLOCKSIZE) / dimBlock.x);
	hipDeviceSynchronize();
	cleap_kernel_paint_mesh<<< dimGrid, dimBlock >>>(dptr, vcount, r, g, b, a);
	hipDeviceSynchronize();
	// unmap buffer object
	hipGraphicsUnmapResources(1, &dm->vbo_c_cuda, 0);
	//printf("ok\n");

	return CLEAP_SUCCESS;
}

CLEAP_RESULT cleap_render_mesh(_cleap_mesh *m){

	if(m->status == CLEAP_SUCCESS && m->dm->status == CLEAP_SUCCESS){
	  	glEnable (GL_POLYGON_OFFSET_FILL); 	//Necesario para permitir dibujar 2 poligonos
    		glPolygonOffset (1.0, 1.0); 		//coplanares (Wireframe y poligono solido)
		glEnable(GL_BLEND);
		glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
		glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, m->dm->eab);

		//! position vectors
		glBindBuffer(GL_ARRAY_BUFFER, m->dm->vbo_v);
		glEnableClientState(GL_VERTEX_ARRAY);
		glVertexPointer(3,      GL_FLOAT, 4*sizeof(float), 0);
		//! normal vectors
		glBindBuffer(GL_ARRAY_BUFFER, m->dm->vbo_n);
		glEnableClientState(GL_NORMAL_ARRAY);
		glNormalPointer(        GL_FLOAT, 4*sizeof(float), 0);
		//! color vectors
		glBindBuffer(GL_ARRAY_BUFFER, m->dm->vbo_c);
		glEnableClientState(GL_COLOR_ARRAY);
		glColorPointer(4,       GL_FLOAT, 4*sizeof(float), 0);

		if (m->solid){
			glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
			glDrawElements(GL_TRIANGLES, cleap_get_face_count(m)*3, GL_UNSIGNED_INT, BUFFER_OFFSET(0));
		}
		if (m->wireframe){
			glDisableClientState(GL_COLOR_ARRAY);
			glColor3f(0.0f, 0.0f, 1.0f);
			glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
			glDrawElements(GL_TRIANGLES, cleap_get_face_count(m)*3, GL_UNSIGNED_INT, BUFFER_OFFSET(0));
		}
		if (m->circumcenters){ //TESIS
			glBindBuffer(GL_ARRAY_BUFFER, m->dm->circumcenters);
			glVertexPointer(3,      GL_FLOAT, 4*sizeof(float), 0);
			glDisableClientState(GL_COLOR_ARRAY);  
			glEnable(GL_PROGRAM_POINT_SIZE);
			glPointSize(10);
			glColor3f(1.0f, 0.0f, 0.0f);
			glPolygonMode(GL_FRONT_AND_BACK, GL_POINT);
			glDrawElements(GL_POINTS, cleap_get_face_count(m)*3, GL_UNSIGNED_INT, BUFFER_OFFSET(0)); //Indicar numero de objetos
		}

		glBindBuffer(GL_ARRAY_BUFFER, 0);
		glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);
		glDisableClientState(GL_NORMAL_ARRAY);
		glDisableClientState(GL_VERTEX_ARRAY);
		glDisableClientState(GL_COLOR_ARRAY);
		glDisable(GL_POLYGON_OFFSET_FILL);
		glDisable(GL_BLEND);
	}
	return CLEAP_SUCCESS;
	
}

CLEAP_RESULT cleap_sync_mesh(_cleap_mesh *m){

	float4 *d_vbo_v, *d_vbo_n, *d_vbo_c;
	GLuint *d_eab;

	size_t num_bytes=0;
	int mem_size_vbo = cleap_get_vertex_count(m)*sizeof(float4);
	int mem_size_eab = 3*cleap_get_face_count(m)*sizeof(GLuint);
	int mem_size_edges = sizeof(int2)*cleap_get_edge_count(m);

	hipGraphicsMapResources(1, &m->dm->vbo_v_cuda, 0);
	hipGraphicsMapResources(1, &m->dm->vbo_n_cuda, 0);
	hipGraphicsMapResources(1, &m->dm->vbo_c_cuda, 0);
	hipGraphicsMapResources(1, &m->dm->eab_cuda, 0);

	hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_v, &num_bytes, m->dm->vbo_v_cuda);
	hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_n, &num_bytes, m->dm->vbo_v_cuda);
	hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_c, &num_bytes, m->dm->vbo_v_cuda);
	hipGraphicsResourceGetMappedPointer( (void**)&d_eab, &num_bytes, m->dm->eab_cuda);

	hipMemcpy( m->vnc_data.v, d_vbo_v, mem_size_vbo, hipMemcpyDeviceToHost );
	hipMemcpy( m->vnc_data.n, d_vbo_n, mem_size_vbo, hipMemcpyDeviceToHost );
	hipMemcpy( m->vnc_data.c, d_vbo_c, mem_size_vbo, hipMemcpyDeviceToHost );
	hipMemcpy( m->triangles, d_eab, mem_size_eab, hipMemcpyDeviceToHost );

	hipGraphicsUnmapResources(1, &m->dm->vbo_v_cuda, 0);
	hipGraphicsUnmapResources(1, &m->dm->vbo_n_cuda, 0);
	hipGraphicsUnmapResources(1, &m->dm->vbo_c_cuda, 0);
	hipGraphicsUnmapResources(1, &m->dm->eab_cuda, 0);

	hipMemcpy( m->edge_data.n, m->dm->d_edges_n, mem_size_edges, hipMemcpyDeviceToHost );
	hipMemcpy( m->edge_data.a, m->dm->d_edges_a, mem_size_edges, hipMemcpyDeviceToHost );
	hipMemcpy( m->edge_data.b, m->dm->d_edges_b, mem_size_edges, hipMemcpyDeviceToHost );
	hipMemcpy( m->edge_data.op, m->dm->d_edges_op, mem_size_edges, hipMemcpyDeviceToHost );

	return CLEAP_SUCCESS;

}

void cleap_print_mesh( _cleap_mesh *m ){

	cleap_sync_mesh(m);
	float4 *d_vbo_v, *d_vbo_n, *d_vbo_c;
	GLuint *d_eab;
	float4 *h_vbo_v, *h_vbo_n, *h_vbo_c;
	GLuint *h_eab;

	h_vbo_v = (float4*)malloc(cleap_get_vertex_count(m)*sizeof(float4));
	h_vbo_n = (float4*)malloc(cleap_get_vertex_count(m)*sizeof(float4));
	h_vbo_c = (float4*)malloc(cleap_get_vertex_count(m)*sizeof(float4));
	h_eab = (GLuint*)malloc(3*cleap_get_face_count(m)*sizeof(GLuint));

	size_t num_bytes=0;
	int mem_size_vbo = cleap_get_vertex_count(m)*sizeof(float4);
	int mem_size_eab = 3*cleap_get_face_count(m)*sizeof(GLuint);

	hipGraphicsMapResources(1, &m->dm->vbo_v_cuda, 0);
	hipGraphicsMapResources(1, &m->dm->vbo_n_cuda, 0);
	hipGraphicsMapResources(1, &m->dm->vbo_c_cuda, 0);
	hipGraphicsMapResources(1, &m->dm->eab_cuda, 0);

	hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_v, &num_bytes, m->dm->vbo_v_cuda);
	hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_n, &num_bytes, m->dm->vbo_v_cuda);
	hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_c, &num_bytes, m->dm->vbo_v_cuda);

	hipGraphicsResourceGetMappedPointer( (void**)&d_eab, &num_bytes, m->dm->eab_cuda);

	hipMemcpy( h_vbo_v, d_vbo_v, mem_size_vbo, hipMemcpyDeviceToHost );
	hipMemcpy( h_vbo_n, d_vbo_n, mem_size_vbo, hipMemcpyDeviceToHost );
	hipMemcpy( h_vbo_c, d_vbo_c, mem_size_vbo, hipMemcpyDeviceToHost );

	hipMemcpy( h_eab, d_eab, mem_size_eab, hipMemcpyDeviceToHost );

	hipGraphicsUnmapResources(1, &m->dm->vbo_v_cuda, 0);
	hipGraphicsUnmapResources(1, &m->dm->vbo_n_cuda, 0);
	hipGraphicsUnmapResources(1, &m->dm->vbo_c_cuda, 0);
	hipGraphicsUnmapResources(1, &m->dm->eab_cuda, 0);

	for(int i=0; i<cleap_get_vertex_count(m); i++){
		printf("mesh_data[%i] = (%f, %f, %f)  w=%f\n", i, h_vbo_v[i].x, h_vbo_v[i].y, h_vbo_v[i].z, h_vbo_v[i].w);
	}
	for(int i=0; i<cleap_get_face_count(m); i++){
		printf("T[%i] = (%i, %i, %i)\n", i, h_eab[3*i], h_eab[3*i+1], h_eab[3*i+2]);
	}

	for( int i=0; i<cleap_get_edge_count(m); i++ ){
	    printf("edge[%i]:\n", i);
	    printf("\tn = (%i, %i)\t", m->edge_data.n[i].x, m->edge_data.n[i].y);
	    printf("a = (%i, %i)\t", m->edge_data.a[i].x, m->edge_data.a[i].y);
	    printf("b = (%i, %i)\n", m->edge_data.b[i].x, m->edge_data.b[i].y);
	}
}

CLEAP_RESULT cleap_delaunay_transformation(_cleap_mesh *m, int mode){

	//printf("CLEAP::delaunay_transformation_%id::", mode);
	float4 *d_vbo_v;
	GLuint *d_eab;
	size_t bytes=0;
	int *h_listo, it=0;
	// Map resources
	hipGraphicsMapResources(1, &m->dm->vbo_v_cuda, 0);
	hipGraphicsMapResources(1, &m->dm->eab_cuda, 0);
	hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_v, &bytes, m->dm->vbo_v_cuda);
	hipGraphicsResourceGetMappedPointer( (void**)&d_eab, &bytes, m->dm->eab_cuda);
	// TEXTURE
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<GLuint>();
	hipBindTexture(0, tex_triangles, d_eab, channelDesc, cleap_get_face_count(m)*3*sizeof(GLuint));
	int block_size = CLEAP_CUDA_BLOCKSIZE;
	dim3 dimBlock(block_size);
	dim3 dimGrid((cleap_get_edge_count(m)+block_size-1) / dimBlock.x);
	dim3 dimBlockInit(block_size);
	dim3 dimGridInit((cleap_get_face_count(m)+block_size-1) / dimBlock.x);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	// if C.C is 1.2 or higher, then use zero-copy for the flag
	if( (deviceProp.major == 1 && deviceProp.minor >= 2) || (deviceProp.major >= 2) ){
		//printf("CLEAP::device::gpu::%s\n", deviceProp.name );
		//printf("CLEAP::device_property::canMapHostMemory = %i\n", deviceProp.canMapHostMemory);
		hipHostAlloc((void **)&h_listo, sizeof(int), hipHostMallocMapped);
		h_listo[0] = 0;
		hipHostGetDevicePointer((void **)&m->dm->d_listo, (void *)h_listo, 0);
		_cleap_start_timer();
		while( !h_listo[0] ){
			h_listo[0] = 1;
			hipDeviceSynchronize();
			_cleap_init_device_dual_arrays_int(m->dm->d_trirel, m->dm->d_trireservs, cleap_get_face_count(m), -1, dimBlockInit, dimGridInit); //demora el orden de 10^-5 secs
			hipDeviceSynchronize();
			if( mode == CLEAP_MODE_2D )
				cleap_kernel_exclusion_processing_2d<256><<< dimGrid, dimBlock >>>(d_vbo_v, d_eab, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m), m->dm->d_listo, m->dm->d_trirel, m->dm->d_trireservs);
			else 
				cleap_kernel_exclusion_processing_3d<256><<< dimGrid, dimBlock >>>(d_vbo_v, d_eab, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m), m->dm->d_listo, m->dm->d_trirel, m->dm->d_trireservs);
			
			hipDeviceSynchronize();
			if( h_listo[0] ){break;}
			cleap_kernel_repair<<< dimGrid, dimBlock >>>(d_eab, m->dm->d_trirel, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m)); //update
			it++;
		}
	}
	// else use memcpy transfers
	else{
		//! ZERO COPY = OFF
		//printf("CLEAP::device::gpu::%s\n", deviceProp.name );
		h_listo = (int*)malloc(sizeof(int));
		h_listo[0] = 0;
		hipMalloc( (void**) &m->dm->d_listo , sizeof(int) );
		//listo es una variable que indica cuando el algoritmo ha finalizado. cuanto listo = 1 entonces todos los edges son delaunay.
		_cleap_start_timer();
		while( !h_listo[0] ){

			h_listo[0] = 1;
			hipMemcpy( m->dm->d_listo, h_listo, sizeof(int), hipMemcpyHostToDevice );
			_cleap_init_device_dual_arrays_int(m->dm->d_trirel, m->dm->d_trireservs, cleap_get_face_count(m), -1, dimBlockInit, dimGridInit); //demora el orden de 10^-5 secs
			if( mode == CLEAP_MODE_2D )
				cleap_kernel_exclusion_processing_2d<256><<< dimGrid, dimBlock >>>(d_vbo_v, d_eab, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m), m->dm->d_listo, m->dm->d_trirel, m->dm->d_trireservs);
			else 
				cleap_kernel_exclusion_processing_3d<256><<< dimGrid, dimBlock >>>(d_vbo_v, d_eab, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m), m->dm->d_listo, m->dm->d_trirel, m->dm->d_trireservs);
			
			hipDeviceSynchronize();
			hipMemcpy( h_listo, m->dm->d_listo, sizeof(int), hipMemcpyDeviceToHost );
			if( h_listo[0] ){
				break;
			}
			cleap_kernel_repair<<< dimGrid, dimBlock >>>(d_eab, m->dm->d_trirel, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m)); //update
			it++;
		}
		hipFree(m->dm->d_listo);
	}
	//printf("computed in %.5g[s] (%i iterations)\n", _cleap_stop_timer(), it );
	//printf("%.6f\n", _cleap_stop_timer());
	//!Unbind Texture
	hipUnbindTexture(tex_triangles);
	// unmap buffer object
	hipGraphicsUnmapResources(1, &m->dm->vbo_v_cuda, 0);
	hipGraphicsUnmapResources(1, &m->dm->eab_cuda, 0);
	hipHostFree(h_listo);

    cleap_sync_mesh(m);
    cleap_calculating_cirucumcenter_2D(m);
	return CLEAP_SUCCESS;

}

int cleap_delaunay_transformation_interactive(_cleap_mesh *m, int mode){

	float4 *d_vbo_v;
	GLuint *d_eab;
	size_t bytes=0;
	int *h_listo, it=0, *flips;
/*/
	fprintf(stdout,"triangulo 1> %i,%i,%i\n", m->triangles[0], m->triangles[1], m->triangles[2]);
	fprintf(stdout,"triangulo 2> %i,%i,%i\n", m->triangles[3], m->triangles[4], m->triangles[5]);
	fprintf(stdout,"triangulo 3> %i,%i,%i\n", m->triangles[6], m->triangles[7], m->triangles[8]);
	fprintf(stdout,"triangulo 4> %i,%i,%i\n", m->triangles[9], m->triangles[10], m->triangles[11]);/*/
	hipGraphicsMapResources(1, &m->dm->vbo_v_cuda, 0);
	hipGraphicsMapResources(1, &m->dm->eab_cuda, 0);
	hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_v, &bytes, m->dm->vbo_v_cuda);
	hipGraphicsResourceGetMappedPointer( (void**)&d_eab, &bytes, m->dm->eab_cuda);

	// TEXTURE
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<GLuint>();
	hipBindTexture(0, tex_triangles, d_eab, channelDesc, cleap_get_face_count(m)*3*sizeof(GLuint));
	int block_size = CLEAP_CUDA_BLOCKSIZE;
	dim3 dimBlock(block_size);
	dim3 dimGrid((cleap_get_edge_count(m)+block_size-1) / dimBlock.x);
	dim3 dimBlockInit(block_size);
	dim3 dimGridInit((cleap_get_face_count(m)+block_size-1) / dimBlock.x);
	hipHostAlloc((void **)&h_listo, sizeof(int), hipHostMallocMapped);
	hipHostAlloc((void **)&flips, sizeof(int), hipHostMallocMapped);
	h_listo[0] = 0;
	flips[0] = 0;
	int *dflips;
	hipHostGetDevicePointer((void **)&m->dm->d_listo, (void *)h_listo, 0);
	hipHostGetDevicePointer((void **)&dflips, (void *)flips, 0);		
	_cleap_start_timer();
	// compute iteration
	h_listo[0] = 1;
	hipDeviceSynchronize();
	_cleap_init_device_dual_arrays_int(m->dm->d_trirel, m->dm->d_trireservs, cleap_get_face_count(m), -1, dimBlockInit, dimGridInit); //demora el orden de 10^-5 secs
	hipDeviceSynchronize();
	if( mode == CLEAP_MODE_2D )
		cleap_kernel_exclusion_processing_2d_debug<256><<< dimGrid, dimBlock >>>(d_vbo_v, d_eab, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m), m->dm->d_listo, m->dm->d_trirel, m->dm->d_trireservs, dflips);
	else 
		cleap_kernel_exclusion_processing_3d<256><<< dimGrid, dimBlock >>>(d_vbo_v, d_eab, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m), m->dm->d_listo, m->dm->d_trirel, m->dm->d_trireservs);
	
	hipDeviceSynchronize();
	if( h_listo[0] ){
		hipUnbindTexture(tex_triangles);
		// unmap buffer object
		hipGraphicsUnmapResources(1, &m->dm->vbo_v_cuda, 0);
		hipGraphicsUnmapResources(1, &m->dm->eab_cuda, 0);
		hipHostFree(h_listo);
		return 0;
	}
	cleap_kernel_repair<<< dimGrid, dimBlock >>>(d_eab, m->dm->d_trirel, m->dm->d_edges_n, m->dm->d_edges_a, m->dm->d_edges_b, m->dm->d_edges_op, cleap_get_edge_count(m)); //update
	it++;
	//printf("CLEAP::delaunay_transformation_%id:: Iteration computed in %.5g[s]\n", mode, _cleap_stop_timer() );
	//!Unbind Texture
	hipUnbindTexture(tex_triangles);
	// unmap buffer object
	hipGraphicsUnmapResources(1, &m->dm->vbo_v_cuda, 0);
	hipGraphicsUnmapResources(1, &m->dm->eab_cuda, 0);
	hipHostFree(h_listo);

	cleap_sync_mesh(m);
	cleap_calculating_cirucumcenter_2D(m);
	return *flips;

}
CLEAP_RESULT cleap_clear_mesh(_cleap_mesh *m){

	if(m->status){
		free(m->vnc_data.v);
		free(m->vnc_data.n);
		free(m->vnc_data.c);
		free(m->edge_data.n);
		free(m->edge_data.a);
		free(m->edge_data.b);
		free(m->edge_data.op);
		free(m->triangles);
		free(m->circumcenters_data);

		if(m->dm->status){
			hipFree(m->dm->d_edges_n);
			hipFree(m->dm->d_edges_a);
			hipFree(m->dm->d_edges_b);
			hipFree(m->dm->d_edges_op);

			hipFree(m->dm->d_trirel);
			hipFree(m->dm->d_trireservs);
			hipFree(m->dm->d_listo);

			glDeleteBuffers(1, &m->dm->vbo_v );
			glDeleteBuffers(1, &m->dm->vbo_n );
			glDeleteBuffers(1, &m->dm->vbo_c );
			glDeleteBuffers(1, &m->dm->eab );

			// opengl method above should have deleted the arrays, this following lines are the equivalent on cuda
			float4 *d_vbo_v, *d_vbo_n, *d_vbo_c;
			GLuint *d_eab;
			size_t bytes=0;

			hipGraphicsMapResources(1, &m->dm->vbo_v_cuda, 0);
			hipGraphicsMapResources(1, &m->dm->vbo_n_cuda, 0);
			hipGraphicsMapResources(1, &m->dm->vbo_c_cuda, 0);
			hipGraphicsMapResources(1, &m->dm->eab_cuda, 0);
			hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_v, &bytes, m->dm->vbo_v_cuda);
			hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_n, &bytes, m->dm->vbo_n_cuda);
			hipGraphicsResourceGetMappedPointer( (void**)&d_vbo_c, &bytes, m->dm->vbo_c_cuda);
			hipGraphicsResourceGetMappedPointer( (void**)&d_eab, &bytes, m->dm->eab_cuda);

			hipFree(d_vbo_v);
			hipFree(d_vbo_n);
			hipFree(d_vbo_c);
			hipFree(d_eab);

		}
		delete m->dm;
		delete m;
	}
	//printf("CLEAP::clear_mesh::");
	//_cleap_print_gpu_mem();
	return CLEAP_SUCCESS;
}

CLEAP_RESULT cleap_save_mesh(_cleap_mesh *m, const char *filename){

	//before saving mesh, we have to sync the data from device and host
	cleap_sync_mesh(m);
	int vcount = cleap_get_vertex_count(m);
	int fcount = cleap_get_face_count(m);
	int ecount = cleap_get_edge_count(m);
	//following line is for computer with other languages.
	setlocale(LC_NUMERIC, "POSIX");
	FILE *file_descriptor = fopen(filename,"w");
	fprintf(file_descriptor,"OFF\n");
	fprintf(file_descriptor,"%d %d %d\n",vcount, fcount, ecount);
	for(int i=0; i<vcount; i++) {
		fprintf(file_descriptor,"%f %f %f\n",m->vnc_data.v[i].x,m->vnc_data.v[i].y,m->vnc_data.v[i].z);
	}
	for(int i=0; i<fcount; i++) {
		fprintf(file_descriptor,"%d %d %d %d\n", 3, m->triangles[i*3+0],m->triangles[i*3+1], m->triangles[i*3+2] );
	}
	fclose(file_descriptor);
	setlocale(LC_NUMERIC, "");
	return CLEAP_SUCCESS;
}

CLEAP_RESULT cleap_save_mesh_no_sync(_cleap_mesh *m, const char *filename){

	int vcount = cleap_get_vertex_count(m);
	int fcount = cleap_get_face_count(m);
	int ecount = cleap_get_edge_count(m);
	//following line is for computer with other languages.
	setlocale(LC_NUMERIC, "POSIX");
	FILE *file_descriptor = fopen(filename,"w");
	fprintf(file_descriptor,"OFF\n");
	fprintf(file_descriptor,"%d %d %d\n",vcount, fcount, ecount);
	for(int i=0; i<vcount; i++) {
		fprintf(file_descriptor,"%f %f %f\n",m->vnc_data.v[i].x,m->vnc_data.v[i].y,m->vnc_data.v[i].z);
	}
	for(int i=0; i<fcount; i++) {
		fprintf(file_descriptor,"%d %d %d %d\n", 3, m->triangles[i*3+0],m->triangles[i*3+1], m->triangles[i*3+2] );
	}
	fclose(file_descriptor);
	setlocale(LC_NUMERIC, "");
	return CLEAP_SUCCESS;
}

void _cleap_start_timer(){
    gettimeofday(&t_ini, NULL); //Tiempo de Inicio
}
double _cleap_stop_timer(){
    gettimeofday(&t_fin, NULL); //Tiempo de Termino
    return (double)(t_fin.tv_sec + (double)t_fin.tv_usec/1000000) - (double)(t_ini.tv_sec + (double)t_ini.tv_usec/1000000);
}

void _cleap_reset_minmax(_cleap_mesh* m){

	m->min_coords.x = FLT_MAX;
	m->min_coords.y = FLT_MAX;
	m->min_coords.z = FLT_MAX;
	m->max_coords.x = -1*FLT_MAX;
	m->max_coords.y = -1*FLT_MAX;
	m->max_coords.z = -1*FLT_MAX;
}

CLEAP_RESULT _cleap_normalize_normals(_cleap_mesh *m){

	//printf("CLEAP::kernel::normalize_normals::");
	size_t bytes;
	float4 *dptr;
	int vcount = cleap_get_vertex_count(m);
	cleap_device_mesh *dm = m->dm;
	hipGraphicsMapResources(1, &dm->vbo_n_cuda, 0);
	hipGraphicsResourceGetMappedPointer((void**)&dptr, &bytes, dm->vbo_n_cuda );

	dim3 dimBlock(CLEAP_CUDA_BLOCKSIZE);
	dim3 dimGrid( (vcount+CLEAP_CUDA_BLOCKSIZE)/dimBlock.x);
	hipDeviceSynchronize();
	cleap_kernel_normalize_normals<<< dimGrid, dimBlock >>>(dptr, vcount);
	hipDeviceSynchronize();
	// unmap buffer object
	hipGraphicsUnmapResources(1, &dm->vbo_n_cuda, 0);
	//printf("ok\n");

	return CLEAP_SUCCESS;
}


CLEAP_RESULT _cleap_device_load_mesh(_cleap_mesh* m){

	// CLEAP::DEVICE_LOAD:: create instance of device_mesh struct
	m->dm = new cleap_device_mesh();
	cleap_device_mesh *dmesh = m->dm;
	hipError_t err;
	// CLEAP::DEVICE_LOAD:: get sizes of _cleap_mesh arrays, in bytes
	GLintptr size = cleap_get_vertex_count(m) *4* sizeof(float);
	GLintptr triangles_bytes_size = sizeof(GLuint)*cleap_get_face_count(m)*3;

	// CLEAP::DEVICE_LOAD:: vbo vertex data
	glGenBuffers(1, &dmesh->vbo_v);
	glBindBuffer(GL_ARRAY_BUFFER, dmesh->vbo_v);
	glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
	glBufferSubData(GL_ARRAY_BUFFER, 0, size, m->vnc_data.v);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	err = hipGraphicsGLRegisterBuffer(&dmesh->vbo_v_cuda, dmesh->vbo_v, cudaGraphicsMapFlagsNone);
	if( err != hipSuccess )
		printf("CLEAP::device_load_mesh::cudaGraphicsRegisterBuffer::vbo_p:: %s\n", hipGetErrorString(err));
	
	// CLEAP::DEVICE_LOAD:: vbo normal data
	glGenBuffers(1, &dmesh->vbo_n);
	glBindBuffer(GL_ARRAY_BUFFER, dmesh->vbo_n);
	glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
	glBufferSubData(GL_ARRAY_BUFFER, 0, size, m->vnc_data.n);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	err = hipGraphicsGLRegisterBuffer(&dmesh->vbo_n_cuda, dmesh->vbo_n, cudaGraphicsMapFlagsNone);
	if( err != hipSuccess )	
		printf("CLEAP::device_load_mesh::cudaGraphicsRegisterBuffer::vbo_n:: %s\n", hipGetErrorString(err));

	// CLEAP::DEVICE_LOAD:: vbo color data
	glGenBuffers(1, &dmesh->vbo_c);
	glBindBuffer(GL_ARRAY_BUFFER, dmesh->vbo_c);
	glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
	glBufferSubData(GL_ARRAY_BUFFER, 0, size, m->vnc_data.c);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	err = hipGraphicsGLRegisterBuffer(&dmesh->vbo_c_cuda, dmesh->vbo_c, cudaGraphicsMapFlagsNone);
	if( err != hipSuccess )
		printf("CLEAP::device_load_mesh::cudaGraphicsRegisterBuffer::vbo_c:: %s\n", hipGetErrorString(err));
	

	// CLEAP::DEVICE_LOAD:: eab data
	glGenBuffers(1, &dmesh->eab);                                                                                   // Generate buffer //index VBO
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, dmesh->eab);                                                             // Bind the element array buffer
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, triangles_bytes_size , 0, GL_STATIC_DRAW);
	glBufferSubData(GL_ELEMENT_ARRAY_BUFFER, 0, triangles_bytes_size, m->triangles);                                 //llenar indices por OpenGL -- OPCION A
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);
	err = hipGraphicsGLRegisterBuffer(&dmesh->eab_cuda, dmesh->eab, cudaGraphicsMapFlagsNone);
	if( err != hipSuccess )
		printf("CLEAP::device_load_mesh::cudaGraphicsRegisterBuffer::eab:: %s\n", hipGetErrorString(err));

	// CLEAP::DEVICE_LOAD:: edges data
	// CLEAP::DEVICE_LOAD:: malloc mesh and aux arrays
	size_t edge_bytes_size  = sizeof(int2)* cleap_get_edge_count(m);
	size_t face_bytes_size = sizeof(int)*cleap_get_face_count(m);
	hipMalloc( (void**) &dmesh->d_edges_n , edge_bytes_size );
	hipMalloc( (void**) &dmesh->d_edges_a , edge_bytes_size );
	hipMalloc( (void**) &dmesh->d_edges_b , edge_bytes_size );
	hipMalloc( (void**) &dmesh->d_edges_op , edge_bytes_size );
	hipMalloc( (void**) &dmesh->d_trirel, face_bytes_size );
	hipMalloc( (void**) &dmesh->d_trireservs, face_bytes_size );

	// CLEAP::DEVICE_LOAD:: memcpy mesh and aux arrays
	hipMemcpy( dmesh->d_edges_n, m->edge_data.n , edge_bytes_size, hipMemcpyHostToDevice );
	hipMemcpy( dmesh->d_edges_a, m->edge_data.a , edge_bytes_size, hipMemcpyHostToDevice );
	hipMemcpy( dmesh->d_edges_b, m->edge_data.b , edge_bytes_size, hipMemcpyHostToDevice );
	hipMemcpy( dmesh->d_edges_op, m->edge_data.op , edge_bytes_size, hipMemcpyHostToDevice );

	// CLEAP::DEVICE_LOAD:: add new device mesh entry into the array of device meshes
	// CLEAP::DEVICE_LOAD:: link main mesh with device_mesh id;
	dmesh->status = CLEAP_SUCCESS;
	//printf("CLEAP::device_load_mesh::ok\n");
	//printf("\n");
	fflush(stdout);

	// CLEAP::DEVICE_LOAD:: paint mesh (green by default)
	cleap_paint_mesh(m, 0.0f, 1.0f, 0.0f, 1.0f );

	// CLEAP::DEVICE_LOAD:: normalize normals
	_cleap_normalize_normals(m);

	// CLEAP::DEVICE_LOAD:: print gpu memory
	//printf("CLEAP::");
	//_cleap_print_gpu_mem();

	return CLEAP_SUCCESS;
}


void _cleap_init_array_int(int* h_array, int size, int value){

	int *d_array;
	hipMalloc( (void**) &d_array , size*sizeof(int));
	dim3 dimBlock(CLEAP_CUDA_BLOCKSIZE);
	dim3 dimGrid((size+CLEAP_CUDA_BLOCKSIZE) / dimBlock.x);
	hipDeviceSynchronize();
	cleap_kernel_init_array_int<<< dimGrid, dimBlock >>>(d_array, size, value);
	hipDeviceSynchronize();
	//copy results to host
	hipMemcpy( h_array, d_array, size*sizeof(int), hipMemcpyDeviceToHost );
	hipFree(d_array);
}




void _cleap_init_device_array_int(int* d_array, int length, int value){

	dim3 dimBlock(CLEAP_CUDA_BLOCKSIZE);
	dim3 dimGrid((length+CLEAP_CUDA_BLOCKSIZE) / dimBlock.x);
	hipDeviceSynchronize();
	cleap_kernel_init_array_int<<< dimGrid, dimBlock >>>(d_array, length, value);
	hipDeviceSynchronize();
}

void _cleap_init_device_dual_arrays_int(int* d_array1, int* d_array2, int length, int value, dim3 &dimBlock, dim3 &dimGrid){
	cleap_kernel_init_device_arrays_dual<<< dimGrid, dimBlock >>>(d_array1, d_array2, length, value);
}

void _cleap_print_gpu_mem(){
	size_t free=0, total=0;
	hipMemGetInfo(&free, &total);
	printf("gpu_memory_used::%iMB (%i%%)\n" , (int)((total - free)/(1024*1024)), (int)((float)(total - free)/((float)total)*100.0));
}

int _cleap_choose_best_gpu_id(){

	int num_devices, device, max_device = 0;
	hipGetDeviceCount(&num_devices);
	if (num_devices > 1) {
	      int max_multiprocessors = 0;
	      for (device = 0; device < num_devices; device++) {
		      hipDeviceProp_t properties;
		      hipGetDeviceProperties(&properties, device);
		      if (max_multiprocessors < properties.multiProcessorCount) {
		              max_multiprocessors = properties.multiProcessorCount;
		              max_device = device;
		      }
	      }
	}
	return max_device;
}

void _cleap_print_splash(){

	printf("\n\n************************************************\n");
	printf("****************** cleap-%d.%d.%d *****************\n", CLEAP_VERSION_MAJOR, CLEAP_VERSION_MINOR, CLEAP_VERSION_PATCH);
	printf("************************************************\n");
	printf("			by %s\n\n\n", CLEAP_AUTHOR);
	fflush(stdout);
}

void _cleap_init_cuda(){

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	hipError_t err = cudaGLSetGLDevice( _cleap_choose_best_gpu_id() );
	//printf("CLEAP::init::CudaGLSetGLDevice::%s\n", hipGetErrorString(err));
	//printf("CLEAP::init::gpu::%s\n", deviceProp.name );
	//printf("CLEAP::init::"); _cleap_print_gpu_mem();
	//printf("\n");
}

CLEAP_RESULT _cleap_init_glew(){
	if( glewInit() != GLEW_OK ){
		printf( ">> CLEAP::Init::GLEW Cannot Init\n");
		return CLEAP_FAILURE;
	}
	return CLEAP_SUCCESS;
}

//TESIS

float* res = (float*) malloc(sizeof(float)*3);
float modulo (float* i, float* j){
	//printf("Modulo\n");
	return pow(sqrt(pow(i[0] - j[0], 2) + pow(i[1] - j[1], 2) + pow(i[2] - j[2], 2)),2);
}

float determinante (float a, float b, float c, float d){
	//printf("Det\n");	
	return (a*d) - (b*c);
}

float productoPunto (float* a, float* b){
	return (a[0]*b[0]) + (a[1]*b[1]) + (a[2]*b[2]);
}

int productoCruz (float* a, float* b){
	//printf("Cruz\n");
	res[0] = determinante(a[1], a[2], b[1], b[2]);
	res[1] = -determinante(a[0], a[2], b[0], b[2]);
	res[2] = determinante(a[0], a[1], b[0], b[1]);
	return 0;
}

int mult (float* a, float b){
	//printf("Mult\n");
	res[0] = a[0] * b;
	res[1] = a[1] * b;
	res[2] = a[2] * b;
	return 0;
}

int resta (float* a, float* b){
	//printf("Resta\n");
	res[0] = a[0] - b[0];
	res[1] = a[1] - b[1];
	res[2] = a[2] - b[2];
	return 0;
}

int suma (float* a, float* b){
	//printf("Suma\n");
	res[0] = a[0] + b[0];
	res[1] = a[1] + b[1];
	res[2] = a[2] + b[2];
	return 0;
}

int div(float* a, float b){
	//printf("Div\n");
	res[0] = a[0] / b;
	res[1] = a[1] / b;
	res[2] = a[2] / b;
	return 0;
}

int circumcenter2 (float* a, float* b, float* c){
	//printf("Circumcenter\n");
	float cero[3] = {0,0,0};
	
	resta(b,a);
	float A[3] = {res[0], res[1], res[2]};

	resta(c,b);
	float B[3] = {res[0], res[1], res[2]};
	
	resta(a,c);
	float C[3] = {res[0], res[1], res[2]};

	productoCruz(A,B);
	float AXB[3] = {res[0], res[1], res[2]};

	productoCruz(C,AXB);
	float CXAXB[3] = {res[0], res[1], res[2]};

	suma(a,c);
	float sumca[3] = {res[0], res[1], res[2]};
	
	div(AXB,2);
	float K[3] = {res[0], res[1], res[2]};
	
	div(sumca, 2);
	float r1[3] =  {res[0], res[1], res[2]};  

	mult(CXAXB, (productoPunto(A,B)/(8 * modulo(K,cero))));
	float r2[3] =  {res[0], res[1], res[2]};

	suma(r1,r2);
/*/
	printf("A: %f, %f, %f\n",A[0], A[1], A[2]);
	printf("B: %f, %f, %f\n",B[0], B[1], B[2]);
	printf("C: %f, %f, %f\n",C[0], C[1], C[2]);
	printf("AXB: %f, %f, %f\n",AXB[0], AXB[1], AXB[2]);
	printf("CXAXB: %f, %f, %f\n",CXAXB[0], CXAXB[1], CXAXB[2]);
	printf("sumca: %f, %f, %f\n",sumca[0], sumca[1], sumca[2]);
	printf("K: %f, %f, %f\n",K[0], K[1], K[2]);
	printf("r1: %f, %f, %f\n",r1[0], r1[1], r1[2]);
	printf("r2: %f, %f, %f\n",r2[0], r2[1], r2[2]);
/*/	
	return 0;
}

int circumcenter (float4 p1, float4 p2, float4 p3){
	//printf("Circumcenter\n");
	float cero[3] = {0,0,0};
	float a[3] = {p1.x,p1.y,p1.z};
	float b[3] = {p2.x,p2.y,p2.z};
	float c[3] = {p3.x,p3.y,p3.z};
	
	resta(b,a);
	float restaBA[3] = {res[0], res[1], res[2]};

	resta(c,a);
	float restaCA[3] = {res[0], res[1], res[2]};

	productoCruz(restaBA,restaCA);
	float BAXCA[3] = {res[0], res[1], res[2]};

	productoCruz(BAXCA, restaBA);
	float BAXCAXBA[3] = {res[0], res[1], res[2]};

	productoCruz(restaCA,BAXCA);
	float CAXBAXCA[3] = {res[0], res[1], res[2]};

	mult(BAXCAXBA, modulo(c,a));
	float r1[3] =  {res[0], res[1], res[2]};  

	mult(CAXBAXCA, modulo(b,a));
	float r2[3] =  {res[0], res[1], res[2]};


	float r3 = 2*modulo(BAXCA, cero);
	
	suma(r1,r2);
	float r4[3] = {res[0], res[1], res[2]};

	div(r4,r3);
	float r[3] = {res[0], res[1], res[2]};

	suma(a,r);
/*/
	printf("restaBA: %f, %f, %f\n",restaBA[0], restaBA[1], restaBA[2]);
	printf("restaCA: %f, %f, %f\n",restaCA[0], restaCA[1], restaCA[2]);
	printf("BAXCA: %f, %f, %f\n",BAXCA[0], BAXCA[1], BAXCA[2]);
	printf("BAXCAXBA: %f, %f, %f\n",BAXCAXBA[0], BAXCAXBA[1], BAXCAXBA[2]);
	printf("CAXBAXCA: %f, %f, %f\n",CAXBAXCA[0], CAXBAXCA[1], CAXBAXCA[2]);
	printf("r1: %f, %f, %f\n",r1[0], r1[1], r1[2]);
	printf("r2: %f, %f, %f\n",r2[0], r2[1], r2[2]);
	printf("r3: %f\n",r3);
	printf("r4: %f, %f, %f\n",r4[0], r4[1], r4[2]);
	printf("r: %f, %f, %f\n",r[0], r[1], r[2]);
/*/	
	return 0;
}

int circumcenter2 (float4 p1, float4 p2, float4 p3){
	//printf("Circumcenter\n");
	float cero[3] = {0,0,0};
	float a[3] = {p1.x,p1.y,p1.z};
	float b[3] = {p2.x,p2.y,p2.z};
	float c[3] = {p3.x,p3.y,p3.z};
	
	resta(b,a);
	float A[3] = {res[0], res[1], res[2]};

	resta(c,b);
	float B[3] = {res[0], res[1], res[2]};
	
	resta(a,c);
	float C[3] = {res[0], res[1], res[2]};

	productoCruz(A,B);
	float AXB[3] = {res[0], res[1], res[2]};

	productoCruz(C,AXB);
	float CXAXB[3] = {res[0], res[1], res[2]};

	suma(a,c);
	float sumca[3] = {res[0], res[1], res[2]};
	
	div(AXB,2);
	float K[3] = {res[0], res[1], res[2]};
	
	div(sumca, 2);
	float r1[3] =  {res[0], res[1], res[2]};  

	mult(CXAXB, (productoPunto(A,B)/(8 * modulo(K,cero))));
	float r2[3] =  {res[0], res[1], res[2]};

	suma(r1,r2);
/*/
	printf("A: %f, %f, %f\n",A[0], A[1], A[2]);
	printf("B: %f, %f, %f\n",B[0], B[1], B[2]);
	printf("C: %f, %f, %f\n",C[0], C[1], C[2]);
	printf("AXB: %f, %f, %f\n",AXB[0], AXB[1], AXB[2]);
	printf("CXAXB: %f, %f, %f\n",CXAXB[0], CXAXB[1], CXAXB[2]);
	printf("sumca: %f, %f, %f\n",sumca[0], sumca[1], sumca[2]);
	printf("K: %f, %f, %f\n",K[0], K[1], K[2]);
	printf("r1: %f, %f, %f\n",r1[0], r1[1], r1[2]);
	printf("r2: %f, %f, %f\n",r2[0], r2[1], r2[2]);
/*/	
	return 0;
}

CLEAP_RESULT cleap_calculating_cirucumcenter_2D(_cleap_mesh *m){

	cleap_device_mesh *dmesh = m->dm;
	hipError_t err;
	GLintptr triangles_bytes_size = cleap_get_face_count(m) * 4 * sizeof(float) ; //sizeof(GLuint)*cleap_get_vertex_count(m); //
	fprintf(stdout, "vertex = %i\n", m->vertex_count);
    int j=0;
	for(int i =0; i<m->face_count; i++){//TESIS: 3D points
		float4 p1 = m->vnc_data.v[m->triangles[i*3]];
		float4 p2 = m->vnc_data.v[m->triangles[i*3+1]];
		float4 p3 = m->vnc_data.v[m->triangles[i*3+2]];
		fprintf(stdout, "P1 X Y Z = %f %f %f\n", p1.x, p1.y, p1.z);
		fprintf(stdout, "P2 X Y Z = %f %f %f\n", p2.x, p2.y, p2.z);
		fprintf(stdout, "P3 X Y Z = %f %f %f\n", p3.x, p3.y, p3.z);
        j++;
/*/
		circumcenter(p1,p2,p3);

		fprintf(stdout, "!X Y Z = %f %f %f\n", res[0], res[1], res[2]);
 /*/
		circumcenter2(p1,p2,p3);

		m->circumcenters_data[i].x = res[0]; 
		m->circumcenters_data[i].y = res[1]; 
		m->circumcenters_data[i].z = res[2]; 
		m->circumcenters_data[i].w = 1.0;
        fprintf(stdout, "!!X Y Z = %f %f %f\n\n", res[0], res[1], res[2]);
	}	
	fprintf(stdout, "Finish = %i\n", m->vertex_count);
    for (int i=0; i<j; i++){
        fprintf(stdout, "circumcenters X Y Z = %f %f %f\n\n", m->circumcenters_data[i].x, m->circumcenters_data[i].y, m->circumcenters_data[i].z);
    }
	glGenBuffers(1, &dmesh->circumcenters);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER,dmesh->circumcenters); 
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, triangles_bytes_size, 0, GL_STATIC_DRAW);
	glBufferSubData(GL_ELEMENT_ARRAY_BUFFER, 0, triangles_bytes_size, m->circumcenters_data);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);
	err = hipGraphicsGLRegisterBuffer(&dmesh->circumcenters_cuda, dmesh->circumcenters, cudaGraphicsMapFlagsNone);
	if( err != hipSuccess )
		printf("CLEAP::circumcenter_calculus::cudaGraphicsRegisterBuffer::circumcenters:: %s\n", hipGetErrorString(err));
	m->circumcenters = 1;
    printf("Size of buffer %i, %i\n", 4 * sizeof(float), (int)triangles_bytes_size);


    return CLEAP_SUCCESS;
}


